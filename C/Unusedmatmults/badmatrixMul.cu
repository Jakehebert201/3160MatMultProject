#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//tensor shit
#include <hipblas.h>
/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A, float *B, int wA, int wB, int blockSize) {
    extern __shared__ float sharedMem[];  // Dynamic shared memory
    float* As = sharedMem;
    float* Bs = &sharedMem[blockSize * blockSize];
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Dynamic shared memory allocation
  extern __shared__ float sharedMem[];
  float* As = sharedMem;
  float* Bs = &sharedMem[BLOCK_SIZE * BLOCK_SIZE];

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty * BLOCK_SIZE + tx] = A[a + wA * ty + tx];
    Bs[ty * BLOCK_SIZE + tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv, const dim3 &dimsA, const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int mem_size_C = dimsB.x * dimsA.y * sizeof(float);
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // Allocate host matrix C
    float *h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    // Create multiple streams
    hipStream_t stream1, stream2, computeStream;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&computeStream, hipStreamNonBlocking));

    // Copy host memory to device asynchronously on separate streams
    checkCudaErrors(
        hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream1));
    checkCudaErrors(
        hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream2));

    // Initialize cuBLAS context and set the compute stream
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    hipblasSetStream(handle, computeStream);

    // Ensure data transfers are complete before starting computation
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Test various block sizes to find the best one
    int blockSizes[] = {16, 32, 64, 128, 256, 512, 1024};
    int numBlockSizes = sizeof(blockSizes) / sizeof(int);
    float bestPerf = 0.0f;
    int bestBlockSize = block_size;  // Start with the initial block_size passed to the function

    for (int i = 0; i < numBlockSizes; ++i) {
        int testBlockSize = blockSizes[i];
        dim3 threads(testBlockSize, testBlockSize);
        dim3 grid((dimsB.x + threads.x - 1) / threads.x, (dimsA.y + threads.y - 1) / threads.y);
        size_t sharedMemSize = 2 * testBlockSize * testBlockSize * sizeof(float);

        // Check the shared memory constraints are not exceeded
        if (testBlockSize * testBlockSize <= 1024) {
            // Execute the kernel with the current block size
            hipEvent_t start, stop;
            checkCudaErrors(hipEventCreate(&start));
            checkCudaErrors(hipEventCreate(&stop));
            checkCudaErrors(hipEventRecord(start, computeStream));

            MatrixMulCUDA<<<grid, threads, sharedMemSize, computeStream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x, testBlockSize);

            checkCudaErrors(hipEventRecord(stop, computeStream));
            checkCudaErrors(hipEventSynchronize(stop));

            float msecTotal = 0.0f;
            checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

            // Compute and print the performance
            float msecPerMatrixMul = msecTotal;
            double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) * static_cast<double>(dimsA.y) * static_cast<double>(dimsB.x);
            double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
            
            if (gigaFlops > bestPerf) {
                bestPerf = gigaFlops;
                bestBlockSize = testBlockSize;
            }

            checkCudaErrors(hipEventDestroy(start));
            checkCudaErrors(hipEventDestroy(stop));
        }
    }

    // Output the best performance and corresponding block size
    printf("Best block size: %d\nBest performance: %.2f GFlop/s\n", bestBlockSize, bestPerf);

    // Copy result from device to host on computeStream
    checkCudaErrors(
        hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, computeStream));
    checkCudaErrors(hipStreamSynchronize(computeStream));

    // Validate and print results
    printf("Checking computed result for correctness: ");
    bool correct = true;
    double eps = 1.e-6;  // machine zero
    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory and resources
    hipblasDestroy(handle);
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    checkCudaErrors(hipStreamDestroy(computeStream));

    return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}




/**
 * Program main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart());
  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
  checkCudaErrors(hipProfilerStop());

  exit(matrix_result);
}
